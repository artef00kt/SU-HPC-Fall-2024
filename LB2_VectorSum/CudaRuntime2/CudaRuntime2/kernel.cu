#include "hip/hip_runtime.h"
﻿

#include <vector>
// #include "hip/hip_runtime.h"
// #include ""
#include <stdio.h>
#include <time.h> 
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

static const int blockSize = 1024;
static const int gridSize = 12;

hipError_t vectorSumWithCUDA(int* res, const int* vector, unsigned int size);
int vectorSumWithCPU(int* res, int* vector, unsigned int size);

__global__ void vectorSumKernel(const int* inArr, int* outArr, unsigned int arraySize)
{
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize; // глобальный индекс потока
    const int gridSize = blockSize * gridDim.x;
    int sum = 0;
    for (int i = gthIdx; i < arraySize; i += gridSize)
        sum += inArr[i];
    __shared__ int shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) {
        if (thIdx < size)
            shArr[thIdx] += shArr[thIdx + size];
        __syncthreads();
    }
    if (thIdx == 0)
        outArr[blockIdx.x] = shArr[0];
}

// функция для генерации псевдослучайного массива
int* createRandArray(int size) {
    const int max = 0;
    const int min = 10;
    int* array = new int[size];
    for (int i = 0; i < size; ++i) {
        array[i] = rand() % (min - max + 1) + max;
    }

    return array;
}

// функция для выводы массива в консоль в виде строки
void printArray(int* a, int size) {
    for (int i = 0; i < size; ++i) {
        std::cout << a[i] << ' ';
    }
    std::cout << std::endl;
}


int main()
{
    unsigned int size;
    std::cout << "Input vector size N" << std::endl << "N: ";
    std::cin >> size;

    // выделение всей нужной памяти и генерация вектора
    int* vector = createRandArray(size);
    int* resultCPU = new int;
    int* resultGPU = new int;

    // вычисление произведения матриц на CPU с замером времени
    clock_t startCPU = clock();
    vectorSumWithCPU(resultCPU, vector, size);
    clock_t endCPU = clock();
    double timeCPU = endCPU - startCPU;

    std::cout << "Time on CPU: " << timeCPU / CLOCKS_PER_SEC * 1000.0 << " msec." << std::endl;
    
    // Add vectors in parallel.
    hipError_t cudaStatus = vectorSumWithCUDA(resultGPU, vector, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    std::cout << "Result CPU:" << *resultCPU << std::endl;
    std::cout << "Result GPU:" << *resultGPU << std::endl;

    delete[] vector;
    delete resultCPU;
    delete resultGPU;

    return 0;
}

int vectorSumWithCPU(int* res, int* vector, unsigned int size) {
    *res = 0;
    for (int i = 0; i < size; ++i) {
        *res += vector[i];
    }
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t vectorSumWithCUDA(int* res, const int* vector, unsigned int size)
{
    int* dev_a = nullptr;
    int* dev_b = nullptr;
    hipError_t cudaStatus;

    hipEvent_t startTime, endTime;
    float time;
    hipEventCreate(&startTime);
    hipEventCreate(&endTime);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, gridSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, vector, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    hipEventRecord(startTime, 0);

    // Launch a kernel on the GPU with one thread for each element.
    dim3 threadsPerBlock(16, 16); // 16 * 16 = 256 threads in block
    dim3 numBlocks(ceil(double(size) / threadsPerBlock.x), ceil(double(size) / threadsPerBlock.y));
    // vectorSumKernel<<<numBlocks, threadsPerBlock>>> (dev_a, dev_b, size);
    vectorSumKernel <<<gridSize, blockSize>>> (dev_a, dev_b, size);
    vectorSumKernel <<<1, blockSize>>> (dev_b, dev_b, gridSize);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    hipEventRecord(endTime, 0);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(res, dev_b, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    hipEventElapsedTime(&time, startTime, endTime);

    std::cout << "Time on GPU: " << time << " msec." << std::endl;

Error:
    hipEventDestroy(startTime);
    hipEventDestroy(endTime);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}